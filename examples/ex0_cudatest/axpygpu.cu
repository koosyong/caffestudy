#include "hip/hip_runtime.h"
#include "axpygpu.h"

__global__ void axpy(float a, float *x, float *y)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    y[i] = a*x[i] + y[i];
}

AXPYGPU::AXPYGPU(int n_block_, int n_thread_, float a_)
    :n_block(n_block_), n_thread(n_thread_), a(a_)
{
    n = n_block * n_thread;
    hipMalloc((void **) &x, n*sizeof(float));
    hipMalloc((void **) &y, n*sizeof(float));
}

void AXPYGPU::compute(float* x_, float* y_, float* z_)
{
    hipMemcpy(x, x_, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, y_, n*sizeof(float), hipMemcpyHostToDevice);

    axpy<<<n_block,n_thread>>>(a,x,y);

    hipMemcpy(z_, y, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}

